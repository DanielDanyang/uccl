#include "hip/hip_runtime.h"
#include <assert.h>
#include <cuda_pipeline.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>

#include <chrono>
#include <tuple>
#include <vector>
#include <thread>

#define cudaCheckErrors(msg)                                        \
    do {                                                            \
        hipError_t __err = hipGetLastError();                     \
        if (__err != hipSuccess) {                                 \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg, \
                    hipGetErrorString(__err), __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n");             \
            exit(1);                                                \
        }                                                           \
    } while (0)

struct GPUSignal {
    volatile uint64_t cmd;
    volatile uint64_t ack;
};


__global__ void gpu_issue_command(GPUSignal *signal, int iterations) {
    int tid = threadIdx.x;
    if (tid == 0) {
        for (int i = 0; i < iterations; i++) {
            unsigned long long start = clock64();
            signal->cmd = i + 1;  

            while (signal->ack != (i+1)) {
                __nanosleep(10);
            }
            unsigned long long end = clock64();
            printf("Command %d issued, acked in %llu cycles\n", i + 1, end - start);
        }
    }
}



void cpu_polling(GPUSignal *signal, int iterations) {
    for (int i = 0; i < iterations; ++i) {
        while (signal->cmd != (i + 1)) {
            std::this_thread::yield();
        }
        signal->ack = i + 1; 
    }
}

// make -j
// CUDA_MODULE_LOADING=EAGER ./gpu_to_cpu_bench
int main() {
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudaCheckErrors("hipStreamCreate failed");

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("clock rate: %d\n", prop.clockRate);

    GPUSignal *signal;
    hipHostAlloc(&signal, sizeof(GPUSignal), hipHostMallocMapped);
    signal->cmd = 0;
    signal->ack = 0;

    int iterations = 1000;
    std::thread cpu_thread(cpu_polling, signal, iterations);

    gpu_issue_command<<<1, 32, 0, stream1>>>(signal, iterations);
    cudaCheckErrors("gpu_issue_command kernel failed");
    hipStreamSynchronize(stream1);
    cudaCheckErrors("hipStreamSynchronize failed");

    cpu_thread.join();

    hipHostFree(signal);
    cudaCheckErrors("hipHostFree failed");
    hipStreamDestroy(stream1);
    cudaCheckErrors("hipStreamDestroy failed");

}
