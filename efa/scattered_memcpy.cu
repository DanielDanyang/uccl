#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "scattered_memcpy.cuh"

__global__ void kernelScatteredMemcpy(uint32_t num_copies,
                                      __grid_constant__ const copy_param_t p) {
    // Total threads in the grid.
    int total_threads = gridDim.x * blockDim.x;
    // Compute our unique global thread id.
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    // Number of threads per copy.
    int threads_per_copy = total_threads / num_copies;

    // Map each thread to a copy.
    int copy_idx = global_id / threads_per_copy;
    if (copy_idx >= num_copies) return;  // In case of rounding

    // Compute local thread index within the group assigned to this copy.
    int local_thread_idx = global_id % threads_per_copy;

    // Retrieve parameters for this copy.
    uint64_t total_size = p.size[copy_idx];
    if (total_size == 0) return;

    char* src_ptr = (char*)p.src[copy_idx];
    char* dst_ptr = (char*)p.dst[copy_idx];

    // Copy 8-byte chunks first (if possible)
    uint64_t num_full = total_size / 8;
    uint64_t* src_u64 = (uint64_t*)src_ptr;
    uint64_t* dst_u64 = (uint64_t*)dst_ptr;

    // Each thread in the group copies its portion of 64-bit words.
    for (uint64_t i = local_thread_idx; i < num_full; i += threads_per_copy) {
        dst_u64[i] = src_u64[i];
    }

    // Handle the remaining tail bytes (if any)
    uint64_t tail_start = num_full * 8;
    // Let only one thread in the copy group (e.g. local_thread_idx == 0) copy
    // the tail.
    if (local_thread_idx == 0) {
        for (uint64_t i = tail_start; i < total_size; i++) {
            dst_ptr[i] = src_ptr[i];
        }
    }
}

void launchScatteredMemcpy(uint32_t num_copies, const copy_param_t* params) {
    // Launch the kernel
    kernelScatteredMemcpy<<<THREAD_BLOCKS, THREADS_PER_BLOCK>>>(num_copies,
                                                                *params);

    // Wait for kernel to complete.
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n",
                hipGetErrorString(err));
    }
}

void launchScatteredMemcpyAsync(uint32_t num_copies, const copy_param_t* params,
                                hipStream_t stream) {
    // Launch the kernel
    kernelScatteredMemcpy<<<THREAD_BLOCKS, THREADS_PER_BLOCK, 0, stream>>>(
        num_copies, *params);
}

int pollScatteredMemcpy(hipStream_t stream) {
    hipError_t err = hipStreamQuery(stream);
    if (err != hipSuccess && err != hipErrorNotReady) {
        fprintf(stderr, "hipStreamQuery failed: %s\n",
                hipGetErrorString(err));
        exit(0);
    }
    return err == hipSuccess;
}
